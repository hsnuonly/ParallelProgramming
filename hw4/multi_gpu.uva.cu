#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define BlockSize 32

const int INF = 1000000000;
void input(char *inFileName);
void output(char *outFileName);

void block_FW(int B,char*);
int ceil(int a, int b);
void cal(char* d,size_t pitch,int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height,hipStream_t stream);
void cpu(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);

unsigned int n, m;	
int *d;

double io_time = 0;
double comp_time = 0;
double mem_time = 0;

int main(int argc, char* argv[]) {
    auto io_beg = std::chrono::high_resolution_clock::now();
	input(argv[1]);
    auto io_end = std::chrono::high_resolution_clock::now();
    io_time += std::chrono::duration<double>(io_end-io_beg).count();

	int B = BlockSize;
	block_FW(B,argv[2]);

    io_beg = std::chrono::high_resolution_clock::now();
	output(argv[2]);
    io_end = std::chrono::high_resolution_clock::now();
    io_time += std::chrono::duration<double>(io_end-io_beg).count();
    std::cout<<n<<"\n";
    std::cout<< comp_time <<" "<<mem_time<<" "<<io_time<<"\n";
	return 0;
}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    int *buf = new int[m*3];
    fread(buf, sizeof(int), 3*m, file);
    hipMallocManaged(&d,sizeof(int)*n*n);

    #pragma omp parallel for
    for (unsigned int i = 0; i < n; ++ i) {
        for (unsigned int j = 0; j < n; ++ j) {
            if (i == j) {
                d[i*n+j] = 0;
            } else {
                d[i*n+j] = INF;
            }
        }
    }

    #pragma omp parallel for
    for (unsigned int i = 0; i < m; ++ i) {
        int pair[3];
        // fread(pair, sizeof(int), 3, file);
        for(int j=0;j<3;j++)
            pair[j]=buf[i*3+j];
        d[pair[0]*n+pair[1]] = pair[2];
    }
    fclose(file);
    delete buf;
}

void output(char *outFileName) {
	FILE *outfile = fopen(outFileName, "w");
	fwrite(d, sizeof(int), n*n, outfile);
    fclose(outfile);
}

int ceil(int a, int b) {
	return (a + b - 1) / b;
}

__global__ void kernel_I(char* d,size_t pitch,int block_x,
    int block_y,int n,int B,int r);
__global__ void kernel_II(char* d,size_t pitch,int block_x,
    int block_y,int n,int B,int r);
__global__ void kernel_III(char* d,size_t pitch,int block_x,
    int block_y,int n,int B,int r);

void block_FW(int B, char* outFileName) {
	int round = ceil(n, B);
    char *device_d = (char*)d;
    size_t pitch = sizeof(int)*n;
    // mem_time += std::chrono::duration<double>(mem_end-mem_beg).count();

    for(int dev=0;dev<2;dev++){
        hipSetDevice(dev);
        hipDeviceEnablePeerAccess(!dev,0);
        // cudaMallocPitch(&device_d[dev],&pitch,sizeof(int)*round*B,round*B);
        // cudaMemcpy2DAsync(device_d[dev],pitch,
        //                     d,sizeof(int)*n,
        //                     sizeof(int)*n,n,cudaMemcpyHostToDevice);
    }
    auto comp_beg = std::chrono::high_resolution_clock::now();
	for (int r = 0; r < round; ++r) {
        dim3 dimBlock(B,B);
        dim3 dimGrid(1,1);
        
        for(int dev=0;dev<2;dev++){
            hipSetDevice(dev);
            hipDeviceSynchronize();
        }
        /* Phase 1*/
        // if(r==0)cudaStreamSynchronize(streams[0]);

        // for (int k = r * B; k < (r+1) * B && k < n; ++k) 
        int dev = r>=round/2;
        hipSetDevice(dev);
        kernel_I  <<<dimGrid,dimBlock,0>>>(device_d,pitch,r,r,n,B,r);
        
        dimGrid = dim3(2,round-1);
        kernel_II <<<dimGrid,dimBlock,0>>>(device_d,pitch,0,r,n,B,r);
        
        for(int dev=0;dev<2;dev++){
            hipSetDevice(dev);
            hipDeviceSynchronize();
        }
        
        hipSetDevice(0);
        dimGrid = dim3(round/2,round-1);
        kernel_III<<<dimGrid,dimBlock,0>>>(device_d,pitch,0,0,n,B,r);
        hipSetDevice(1);
        dimGrid = dim3(round-(round/2),round-1);
        kernel_III<<<dimGrid,dimBlock,0>>>(device_d,pitch,(round/2),0,n,B,r);
        
        for(int dev=0;dev<2;dev++){
            hipSetDevice(dev);
            hipDeviceSynchronize();
        }
        
        // std::cout<< std::chrono::duration<double>(comp_end-comp_beg).count()<<"\n";
    }
    // cudaStreamSynchronize(0);
    
    auto comp_end = std::chrono::high_resolution_clock::now();
    comp_time += std::chrono::duration<double>(comp_end-comp_beg).count();
    // mem_beg = std::chrono::high_resolution_clock::now();
    // cudaMemcpy2D(d,sizeof(int)*n,
    //     device_d,pitch,
    //     sizeof(int)*n,n,cudaMemcpyDeviceToHost);
    
    // mem_end = std::chrono::high_resolution_clock::now();
    // mem_time += std::chrono::duration<double>(mem_end-mem_beg).count();
    // cudaMemcpy(d,device_d,sizeof(int)*n*n,cudaMemcpyDeviceToHost);
}

__device__ inline int gmin(int a,int b){
    return (a>b)*b+(a<=b)*a;
}

__global__ void kernel_I(char* d,size_t pitch,int block_x,
    int block_y,int n,int B, int r){
    __shared__ int d_i_j[BlockSize][BlockSize+1];

    const unsigned int i = block_x*B+threadIdx.x;
    const unsigned int j = block_y*B+threadIdx.y;
    // const int idx = threadIdx.y*blockDim.x*threadIdx.x;

    int* d_i = (int*)(d+pitch*i);

    unsigned int origin_path = i<n&&j<n? __ldg(&d_i[j]) : INF;
    d_i_j[threadIdx.x][threadIdx.y] = origin_path;

    // int* d_k_j = (int*)(d+pitch*k);
    const unsigned int k_max = gmin((r+1) * B,n);
    #pragma unroll
    for (unsigned int k = r * B; k < k_max; ++k) {
        __syncthreads();
        int new_d = d_i_j[threadIdx.x][k-r*B]+d_i_j[k-r*B][threadIdx.y];
        if(d_i_j[threadIdx.x][threadIdx.y]>new_d){
            d_i_j[threadIdx.x][threadIdx.y]=new_d;
        }
    }

    if(origin_path>d_i_j[threadIdx.x][threadIdx.y]&&i<n&&j<n){
        d_i[j]=d_i_j[threadIdx.x][threadIdx.y];
    }
}

__global__ void kernel_III(char* d,size_t pitch,int block_x,
    int block_y,int n,int B,int r){
    __shared__ int d_i_k[BlockSize][BlockSize+1];
    __shared__ int d_k_j[BlockSize][BlockSize+1];

    int i = (block_x+blockIdx.x)>=r?
        (block_x+blockIdx.x+1)*B+threadIdx.x:(block_x+blockIdx.x)*B+threadIdx.x;
    int j = (block_y+blockIdx.y)>=r?
        (block_y+blockIdx.y+1)*B+threadIdx.y:(block_y+blockIdx.y)*B+threadIdx.y;
    // unsigned int i = (block_x+blockIdx.x)*B+threadIdx.x;
    // unsigned int j = (block_y+blockIdx.y)*B+threadIdx.y;


    int* d_i = ((int*)(d+pitch*i));
    int path = i<n&&j<n?  __ldg(&d_i[j]) : INF;
    int origin_path = path;
    if(r*B+threadIdx.y < n && i < n)
        d_i_k[threadIdx.x][threadIdx.y] = __ldg(&d_i[r*B+threadIdx.y]);
    else
        d_i_k[threadIdx.x][threadIdx.y] = INF;
    if(r*B+threadIdx.x < n && j < n)
        d_k_j[threadIdx.x][threadIdx.y] = __ldg(&((int*)(d+pitch*(r*B+threadIdx.x)))[j]);
    else 
        d_k_j[threadIdx.x][threadIdx.y] = INF;

    __syncthreads();
    // const int k_max = gmin((r+1) * B , n);
    const unsigned int k_max = gmin((r+1) * B,n);
    #pragma unroll
    for (unsigned int k = r * B; k < k_max; ++k) {
        int new_path = d_i_k[threadIdx.x][k-r*B]+d_k_j[k-r*B][threadIdx.y];
        if(path>new_path) path = new_path;
    }
    if(origin_path>path&&i<n&&j<n){
        d_i[j]=path;
    }
}
__global__ void kernel_II(char* d,size_t pitch,int block_x,
    int block_y,int n,int B,int r){
    __shared__ int d_i_k[BlockSize][BlockSize+1];
    __shared__ int d_k_j[BlockSize][BlockSize+1];

    unsigned int i, j;
    if(blockIdx.x==0){
        i = r*B + threadIdx.x;
        j = blockIdx.y>=r ? 
            (blockIdx.y+1) * B + threadIdx.y : blockIdx.y * B + threadIdx.y ;
    }
    else{
        i = blockIdx.y>=r ? 
            (blockIdx.y+1) * B + threadIdx.x : blockIdx.y * B + threadIdx.x ;
        j = r*B + threadIdx.y;
    }
    // int i = (block_x+blockIdx.x)>=r?
    //     (block_x+blockIdx.x+1)*B+threadIdx.x:(block_x+blockIdx.x)*B+threadIdx.x;
    // int j = (block_y+blockIdx.y)>=r?
    //     (block_y+blockIdx.y+1)*B+threadIdx.y:(block_y+blockIdx.y)*B+threadIdx.y;
    // int j = (block_y+blockIdx.y)*B+threadIdx.y;


    int* d_i = (int*)(d+pitch*i);
    int path = i<n&&j<n? d_i[j] : INF;
    int origin_path = path;
    d_i_k[threadIdx.x][threadIdx.y] = i < n && r*B+threadIdx.y < n ?  __ldg(&d_i[r*B+threadIdx.y]) : INF;
    d_k_j[threadIdx.x][threadIdx.y] = j < n && r*B+threadIdx.x < n ?  __ldg(&((int*)(d+pitch*(r*B+threadIdx.x)))[j]) : INF;

    __syncthreads();
    const unsigned int k_max = gmin((r+1) * B , n);
    #pragma unroll
    for (unsigned int k = r * B; k < k_max; ++k) {
        // int* d_k = (int*)(d+pitch*k);
        int new_path = d_i_k[threadIdx.x][k-r*B]+d_k_j[k-r*B][threadIdx.y];
        if(path>new_path) path = new_path;
    }
    if(origin_path>path&&i<n&&j<n){
        d_i[j]=path;
    }
}

