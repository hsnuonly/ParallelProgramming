#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define BlockSize 32

const int INF = 1000000000;
void input(char *inFileName);
void output(char *outFileName);

void block_FW(int B,char*);
int ceil(int a, int b);
void cal(char* d,size_t pitch,int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height,hipStream_t stream);
void cpu(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);

int n, m;	
int *d;

double io_time = 0;
double comp_time = 0;
double mem_time = 0;

int main(int argc, char* argv[]) {
    auto io_beg = std::chrono::high_resolution_clock::now();
	input(argv[1]);
    auto io_end = std::chrono::high_resolution_clock::now();
    io_time += std::chrono::duration<double>(io_end-io_beg).count();

	int B = BlockSize;
	block_FW(B,argv[2]);

    io_beg = std::chrono::high_resolution_clock::now();
	output(argv[2]);
    io_end = std::chrono::high_resolution_clock::now();
    io_time += std::chrono::duration<double>(io_end-io_beg).count();

    std::cout<< comp_time <<" "<<mem_time<<" "<<io_time<<"\n";
	return 0;
}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    int *buf = new int[m*3];
    fread(buf, sizeof(int), 3*m, file);
    d = new int[n*n];

    #pragma omp parallel for
    for (int i = 0; i < n; ++ i) {
        for (int j = 0; j < n; ++ j) {
            if (i == j) {
                d[i*n+j] = 0;
            } else {
                d[i*n+j] = INF;
            }
        }
    }

    #pragma omp parallel for
    for (int i = 0; i < m; ++ i) {
        int pair[3];
        // fread(pair, sizeof(int), 3, file);
        for(int j=0;j<3;j++)
            pair[j]=buf[i*3+j];
        d[pair[0]*n+pair[1]] = pair[2];
    }
    fclose(file);
    delete buf;
}

void output(char *outFileName) {
	FILE *outfile = fopen(outFileName, "w");
	fwrite(d, sizeof(int), n*n, outfile);
    fclose(outfile);
}

int ceil(int a, int b) {
	return (a + b - 1) / b;
}

__global__ void kernel_I(char* d,size_t pitch,int block_x,
    int block_y,int n,int B,int r);
__global__ void kernel_II(char* d,size_t pitch,int block_x,
    int block_y,int n,int B,int r);
__global__ void kernel_III(char* d,size_t pitch,int block_x,
    int block_y,int n,int B,int r);

void block_FW(int B, char* outFileName) {
	int round = ceil(n, B);
    char *device_d;
    size_t pitch;
    // cudaMalloc(&device_d,sizeof(int)*n*n);
    // cudaMemcpy(device_d,d,sizeof(int)*n*n,cudaMemcpyHostToDevice);
    auto mem_beg = std::chrono::high_resolution_clock::now();    
    hipMallocPitch(&device_d,&pitch,sizeof(int)*round*B,round*B);
    hipMemcpy2DAsync(device_d,pitch,d,sizeof(int)*n,sizeof(int)*n,n,hipMemcpyHostToDevice);
    auto mem_end = std::chrono::high_resolution_clock::now();
    mem_time += std::chrono::duration<double>(mem_end-mem_beg).count();

    // cudaStream_t streams[4];
    // for(int i=0;i<4;i++)
    //     cudaStreamCreate(&streams[i]);
    // cudaMemcpy2DAsync(device_d,pitch,
    //     d,sizeof(int)*n,sizeof(int)*B,B,cudaMemcpyHostToDevice,streams[0]);
    // if(n>B){
    //     cudaMemcpy2DAsync(device_d+sizeof(int)*B,pitch,
    //         d+B,sizeof(int)*n,sizeof(int)*(n-B),B,cudaMemcpyHostToDevice,streams[1]);
    //     cudaMemcpy2DAsync(device_d+pitch*B,pitch,
    //         d+n*B,sizeof(int)*n,sizeof(int)*B,(n-B),cudaMemcpyHostToDevice,streams[1]);
    //     cudaMemcpy2DAsync(device_d+pitch*B+sizeof(int)*B,pitch,
    //         d+n*B+B,sizeof(int)*n,sizeof(int)*(n-B),(n-B),cudaMemcpyHostToDevice,streams[2]);
    // }
    
    auto comp_beg = std::chrono::high_resolution_clock::now();
	for (int r = 0; r < round; ++r) {
        dim3 dimBlock(B,B);
        dim3 dimGrid(1,1);
        
        /* Phase 1*/
        // if(r==0)cudaStreamSynchronize(streams[0]);
        // for (int k = r * B; k < (r+1) * B && k < n; ++k) 
        kernel_I  <<<dimGrid,dimBlock,0>>>(device_d,pitch,r,r,n,B,r);
        
        dimGrid = dim3(2,round-1);
        kernel_II <<<dimGrid,dimBlock,0>>>(device_d,pitch,0,r,n,B,r);
        
        dimGrid = dim3(round-1,round-1);
        kernel_III<<<dimGrid,dimBlock,0>>>(device_d,pitch,0,0,n,B,r);

        
        // std::cout<< std::chrono::duration<double>(comp_end-comp_beg).count()<<"\n";
    }
    hipStreamSynchronize(0);
    
    auto comp_end = std::chrono::high_resolution_clock::now();
    comp_time += std::chrono::duration<double>(comp_end-comp_beg).count();
    mem_beg = std::chrono::high_resolution_clock::now();
    hipMemcpy2D(d,sizeof(int)*n,
        device_d,pitch,
        sizeof(int)*n,n,hipMemcpyDeviceToHost);
    mem_end = std::chrono::high_resolution_clock::now();
    mem_time += std::chrono::duration<double>(mem_end-mem_beg).count();
    
    // cudaStream_t streams[n];
    // for(int i=0;i<n;i++){
    //     cudaStreamCreate(&streams[i]);
    //     cudaMemcpy2DAsync(d+n*i,sizeof(int)*n,
    //         device_d+pitch*i,pitch,
    //         sizeof(int)*n,1,cudaMemcpyDeviceToHost,streams[i]);
    // }
    // FILE *outfile = fopen(outFileName, "w");
    // for(int i=0;i<n;i++){
    //     cudaStreamSynchronize(streams[i]);
    //     fwrite(d+i*n, sizeof(int), n, outfile);
    // }
    // fclose(outfile);
    
    // cudaMemcpy(d,device_d,sizeof(int)*n*n,cudaMemcpyDeviceToHost);
}

__device__ inline int gmin(int a,int b){
    return (a>b)*b+(a<=b)*a;
}

__global__ void kernel_I(char* d,size_t pitch,int block_x,
    int block_y,int n,int B, int r){
    __shared__ int d_i_j[BlockSize][BlockSize+1];

    const int i = block_x*B+threadIdx.x;
    const int j = block_y*B+threadIdx.y;
    // const int idx = threadIdx.y*blockDim.x*threadIdx.x;

    int* d_i = (int*)(d+pitch*i);

    int origin_path = __ldg(&d_i[j]);
    d_i_j[threadIdx.x][threadIdx.y] = origin_path;

    // int* d_k_j = (int*)(d+pitch*k);
    const int k_max = gmin((r+1) * B,n);
    #pragma unroll
    for (int k = r * B; k < k_max; ++k) {
        __syncthreads();
        int new_d = d_i_j[threadIdx.x][k-r*B]+d_i_j[k-r*B][threadIdx.y];
        if(d_i_j[threadIdx.x][threadIdx.y]>new_d){
            d_i_j[threadIdx.x][threadIdx.y]=new_d;
        }
    }

    if(origin_path>d_i_j[threadIdx.x][threadIdx.y]){
        d_i[j]=d_i_j[threadIdx.x][threadIdx.y];
    }
}

__global__ void kernel_III(char* d,size_t pitch,int block_x,
    int block_y,int n,int B,int r){
    __shared__ int d_i_k[BlockSize][BlockSize+1];
    __shared__ int d_k_j[BlockSize][BlockSize+1];

    int x = threadIdx.x;
    int y = threadIdx.y;

    int i = (block_x+blockIdx.x)>=r?
        (block_x+blockIdx.x+1)*BlockSize+x:(block_x+blockIdx.x)*BlockSize+x;
    int j = (block_y+blockIdx.y)>=r?
        (block_y+blockIdx.y+1)*BlockSize+y:(block_y+blockIdx.y)*BlockSize+y;


    int* d_i = ((int*)(d+pitch*i));
    // int path = i<n&&j<n ? __ldg(&d_i[j]) : INF;
    int path = __ldg(&d_i[j]);
    int origin_path = path;
    // if(r*B+y < n && i < n)
    //     d_i_k[x][y] = __ldg(&d_i[r*B+y]);
    // else
    //     d_i_k[x][y] = INF;
    // if(r*B+x < n && j < n)
    //     d_k_j[x][y] = __ldg(&((int*)(d+pitch*(r*B+x)))[j]);
    // else 
    //     d_k_j[x][y] = INF;
    d_i_k[x][y] = __ldg(&d_i[r*BlockSize+y]);
    d_k_j[x][y] = __ldg(&((int*)(d+pitch*(r*BlockSize+x)))[j]);

    __shared__ int k_max;
    if(x==0&&y==0)k_max = gmin(n-r*B,B);
    __syncthreads();
    // const int k_max = gmin((r+1) * B , n);
    #pragma unroll 32
    for (int k = 0; k < k_max; ++k) { 
        int new_path = d_i_k[x][k]+d_k_j[k][y];
        if(path>new_path) path = new_path;
    }
    if(origin_path>path&&i<n&&j<n){
        d_i[j]=path;
    }
}
__global__ void kernel_II(char* d,size_t pitch,int block_x,
    int block_y,int n,int B,int r){
    __shared__ int d_i_k[BlockSize][BlockSize+1];
    __shared__ int d_k_j[BlockSize][BlockSize+1];

    int i, j;
    if(blockIdx.x==0){
        i = r*B + threadIdx.x;
        j = blockIdx.y>=r ? 
            (blockIdx.y+1) * B + threadIdx.y : blockIdx.y * B + threadIdx.y ;
    }
    else{
        i = blockIdx.y>=r ? 
            (blockIdx.y+1) * B + threadIdx.x : blockIdx.y * B + threadIdx.x ;
        j = r*B + threadIdx.y;
    }
    // int i = (block_x+blockIdx.x)>=r?
    //     (block_x+blockIdx.x+1)*B+threadIdx.x:(block_x+blockIdx.x)*B+threadIdx.x;
    // int j = (block_y+blockIdx.y)>=r?
    //     (block_y+blockIdx.y+1)*B+threadIdx.y:(block_y+blockIdx.y)*B+threadIdx.y;
    // int j = (block_y+blockIdx.y)*B+threadIdx.y;


    int* d_i = (int*)(d+pitch*i);
    int path = i<n&&j<n ? d_i[j] : INF;
    int origin_path = path;
    d_i_k[threadIdx.x][threadIdx.y] = i<n&&r*B+threadIdx.y<n ? __ldg(&d_i[r*B+threadIdx.y]) : INF;
    d_k_j[threadIdx.x][threadIdx.y] = r*B+threadIdx.x<n&&j<n ? __ldg(&((int*)(d+pitch*(r*B+threadIdx.x)))[j]) : INF;

    __syncthreads();
    const int k_max = gmin((r+1) * B , n);
    #pragma unroll
    for (int k = r * B; k < k_max; ++k) {
        // int* d_k = (int*)(d+pitch*k);
        int new_path = d_i_k[threadIdx.x][k-r*B]+d_k_j[k-r*B][threadIdx.y];
        if(path>new_path) path = new_path;
    }
    if(origin_path>path){
        d_i[j]=path;
    }
}

